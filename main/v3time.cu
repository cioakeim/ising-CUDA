/* Script that calculates the median execution times for the random state initialization and 
 * the evolution the Ising model with the V2 implementation. Each (n,k) pair is run multiple 
 * times and the median value of both processes execution is stored in the location specified 
 * from the terminal call. Also from the call, the block length for the random state
 * is specified.
 */
#include <stdio.h>
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#include "isingV3.h"

// Each (n,k) pairs is run this many times.
#define RUNS_PER_SIZE 5

int main(int argc, char** argv){
  if(argc!=3){
    printf("Usage: ./v3time [resultFolder] [InitThreadBlockLength]\n");
    exit(1);
  }
  // Get file 
  FILE* result_file;
  char *file_name=(char*)malloc(100*sizeof(char));
  snprintf(file_name,99,"%s/v3time_%s.txt",argv[1],argv[2]);
  result_file=fopen(file_name,"w");
  if(!result_file){
    printf("Error in opening result file..\n");
    exit(1);
  }
  // Init variables.
  char *G;
  char *G0;
  char *HostG;
  // Range of length.
  int n_min=5000;
  int n_step=10000;
  int n_max=35000;
  // Range of iteration steps.
  int k_min=20;
  int k_step=40;
  int k_max=100;
  // For the median calculation. (Time is in ms)
  float init_times_ms[RUNS_PER_SIZE];
  float iter_times_ms[RUNS_PER_SIZE];
  float init_time_ms,iter_time_ms;
  hipEvent_t start,stop,mid;
  hipEventCreate(&start);
  hipEventCreate(&mid);
  hipEventCreate(&stop);
  // For kernel calls.
  dim3 blockSize,gridSize;
  int blockLength;
  int threadBlockLength=atoi(argv[2]);
  // For error checking.
  hipError_t hipError_t;

  // For all sizes.. 
  for(int n=n_min;n<=n_max;n+=n_step){
    // Allocate needed space.
    allocateGridV3(&G0,n);
    allocateGridV3(&G,n);
    HostG=(char*)malloc(n*n*sizeof(char));
    // For all k..
    for(int k=k_min;k<=k_max;k+=k_step){
      // Test many times..
      for(int run_count=0;run_count<RUNS_PER_SIZE;run_count++){
        // Start counting..
        hipEventRecord(start,0);
        getInitializationDimensionsV3(n,blockSize,gridSize,threadBlockLength);
        initializeRandomGridV3<<<gridSize,blockSize>>>(G0,n,threadBlockLength);
        hipDeviceSynchronize();
        // Error check for random state..
        hipError_t=hipGetLastError();
        if(hipError_t!=hipSuccess){
          printf("Kernel failed at initializeRandomGridV3: %s\n",hipGetErrorString(hipError_t));
          exit(1);
        }
        // Get between time..
        hipEventRecord(mid,0);
        // Evolution..
        getEvolutionDimensionsV3(n,blockSize,gridSize,blockLength);
        evolveIsingGridV3(HostG,G,G0,n,k,blockSize,gridSize,blockLength);
        hipEventRecord(stop,0);
        // Timings gathered..
        hipEventSynchronize(mid);
        hipEventSynchronize(stop);
        // Get time intervals..
        hipEventElapsedTime(&init_time_ms,start,mid);
        hipEventElapsedTime(&iter_time_ms,mid,stop);
        // Store time in buffer.
        init_times_ms[run_count]=init_time_ms;
        iter_times_ms[run_count]=iter_time_ms;
      }
      // Get median for this pair (n,k):
      // (Algorithm is inefficient af but the size is small so I don't care)
      float iter_median_ms=0;
      float init_median_ms=0;
      int init_count,iter_count;
      for(int i=0;i<RUNS_PER_SIZE;i++){
        init_count=0;
        iter_count=0;
        for(int j=0;j<RUNS_PER_SIZE;j++){
          if(init_times_ms[i]>=init_times_ms[j]){
            init_count++;
          }
          if(iter_times_ms[i]>=iter_times_ms[j]){
            iter_count++;
          }
        }
        // Check if median:
        if(init_count==(RUNS_PER_SIZE/2)+1){
          init_median_ms=init_times_ms[i];
        }
        if(iter_count==(RUNS_PER_SIZE/2)+1){
          iter_median_ms=iter_times_ms[i];
        }
      }
      // Median retrieved write to file:
      // [n] [k] [init_median] [iter_median]
      fprintf(result_file,"%d %d %f %f\n",n,k,init_median_ms,iter_median_ms);
    }
    freeGridV3(G);
    freeGridV3(G0);
    free(HostG);
    printf("Size %d done.\n",n);
  }
  printf("Job done. V3 times gathered.\n");
  // Cleanup.
  hipEventDestroy(start);
  hipEventDestroy(mid);
  hipEventDestroy(stop);
  free(file_name);
  fclose(result_file);
  return 0; 
}
