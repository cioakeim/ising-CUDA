/* Script that provides the execution times for v1 of the ising model. */
#include <stdio.h>
#include <stdlib.h> 
#include <string.h>
#include <hip/hip_runtime.h>
#include "isingV1.h"

#define RUNS_PER_SIZE 5

int main(int argc, char** argv){
  if(argc!=2){
    printf("Usage: ./v1time [resultFolder]\n");
    exit(1);
  }
  // Get file 
  FILE* result_file;
  char *file_name=(char*)malloc(100*sizeof(char));
  snprintf(file_name,99,"%s/v1time.txt",argv[1]);
  result_file=fopen(file_name,"w");
  if(!result_file){
    printf("Error in opening result file..\n");
    exit(1);
  }

  // Init variables.
  char **G;
  char **G0;
  int n_min=5000;
  int n_step=10000;
  int n_max=35000;
  int k_min=30;
  int k_step=50;
  int k_max=80;
  float init_times[RUNS_PER_SIZE];
  float iter_times[RUNS_PER_SIZE];
  float init_time,iter_time;
  hipEvent_t start,stop,mid;
  hipEventCreate(&start);
  hipEventCreate(&mid);
  hipEventCreate(&stop);
  dim3 blockSize,gridSize;
  hipError_t hipError_t;

  printf("Block size: %d\n",BLOCK_MAX);
  // For all sizes.. 
  for(int n=n_min;n<=n_max;n+=n_step){
    gridAllocateV1(&G0,n);
    gridAllocateV1(&G,n);
    getDimensionsV1(n,blockSize,gridSize);
    // For all k until k_max
    for(int k=k_min;k<=k_max;k+=k_step){
      // Test many times..
      for(int run_count=0;run_count<RUNS_PER_SIZE;run_count++){
        // Run this..
        hipEventRecord(start,0);
        initRandomV1<<<gridSize,blockSize>>>(G0,n);
        // Error check for initRandom..
        hipError_t=hipGetLastError();
        if(hipError_t!=hipSuccess){
          printf("Kernel failed at initRandom: %s\n",hipGetErrorString(hipError_t));
          exit(1);
        }
        hipDeviceSynchronize();
        hipEventRecord(mid,0);
        isingV1(G,G0,n,k,blockSize,gridSize);
        hipEventRecord(stop,0);
        hipEventSynchronize(mid);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&init_time,start,mid);
        hipEventElapsedTime(&iter_time,mid,stop);
        // Reset events.
        hipEventRecord(start,0);
        hipEventRecord(mid,0);
        hipEventRecord(stop,0);
        // Store time in buffer.
        init_times[run_count]=init_time;
        iter_times[run_count]=iter_time;
      }
      // Get median for this pair (n,k):
      float iter_median=0;
      float init_median=0;
      int init_count,iter_count;
      for(int i=0;i<RUNS_PER_SIZE;i++){
        init_count=0;
        iter_count=0;
        for(int j=0;j<RUNS_PER_SIZE;j++){
          if(init_times[i]>=init_times[j]){
            init_count++;
          }
          if(iter_times[i]>=iter_times[j]){
            iter_count++;
          }
        }
        // Check if median:
        if(init_count==(RUNS_PER_SIZE/2)+1){
          init_median=init_times[i];
        }
        if(iter_count==(RUNS_PER_SIZE/2)+1){
          iter_median=iter_times[i];
        }
      }
      // Median retrieved write to file:
      // [n] [k] [median]
      fprintf(result_file,"%d %d %f %f\n",n,k,init_median,iter_median);
    }
    freeGridV1(G);
    freeGridV1(G0);
    printf("Size %d done.\n",n);
  }
  printf("V1 Timing gathered.\n");
  // Cleanup.
  hipEventDestroy(start);
  hipEventDestroy(mid);
  hipEventDestroy(stop);
  free(file_name);
  fclose(result_file);
  return 0; 
}
