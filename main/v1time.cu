/* Script that provides the execution times for v0 of the ising model. */
#include <stdio.h>
#include <stdlib.h> 
#include <string.h>
#include <hip/hip_runtime.h>
#include "isingV1.h"

#define RUNS_PER_SIZE 3

int main(int argc, char** argv){
  if(argc!=2){
    printf("Usage: ./v0time [resultFolder]\n");
    exit(1);
  }
  // Get file 
  FILE* result_file;
  char *file_name=(char*)malloc(100*sizeof(char));
  snprintf(file_name,99,"%s/v1time.txt",argv[1]);
  result_file=fopen(file_name,"w");
  if(!result_file){
    printf("Error in opening result file..\n");
    exit(1);
  }

  // Init variables.
  char **G;
  char **G0;
  int n_min=200;
  int n_step=200;
  int n_max=2000;
  int k_min=20;
  int k_step=20;
  int k_max=100;
  float times[RUNS_PER_SIZE];
  float run_time;
  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  dim3 blockSize,gridSize;
  hipError_t hipError_t;

  // For all sizes.. 
  for(int n=n_min;n<=n_max;n+=n_step){
    gridAllocateV1(&G0,n);
    gridAllocateV1(&G,n);
    getDimensionsV1(n,blockSize,gridSize);
    // For all k until k_max
    for(int k=k_min;k<=k_max;k+=k_step){
      // Test many times..
      for(int run_count=0;run_count<RUNS_PER_SIZE;run_count++){
        // Run this..
        hipEventRecord(start,0);
        initRandomV1<<<gridSize,blockSize>>>(G0,n);
        // Error check for initRandom..
        hipError_t=hipGetLastError();
        if(hipError_t!=hipSuccess){
          printf("Kernel failed at initRandom: %s\n",hipGetErrorString(hipError_t));
          exit(1);
        }
        hipDeviceSynchronize();
        isingV1(G,G0,n,k,blockSize,gridSize);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&run_time,start,stop);
        // Reset events.
        hipEventRecord(start,0);
        hipEventRecord(stop,0);
        // Store time in buffer.
        times[run_count]=run_time;
      }
      // Get median for this pair (n,k):
      float median=0;
      int count;
      for(int i=0;i<RUNS_PER_SIZE;i++){
        count=0;
        for(int j=0;j<RUNS_PER_SIZE;j++){
          if(times[i]>=times[j]){
            count++;
          }
        }
        // Check if median:
        if(count==(RUNS_PER_SIZE/2)+1){
          median=times[i];
          break;
        }
      }
      // Median retrieved write to file:
      // [n] [k] [median]
      fprintf(result_file,"%d %d %f\n",n,k,median);
    }
    freeGridV1(G);
    freeGridV1(G0);
    printf("Size %d done.\n",n);
  }
  printf("V1 Timing gathered.\n");
  // Cleanup.
  hipEventDestroy(start);
  hipEventDestroy(stop);
  free(file_name);
  fclose(result_file);
  return 0; 
}
