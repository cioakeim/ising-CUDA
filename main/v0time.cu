/* Script that provides the execution times for v0 of the ising model. */
#include <stdio.h>
#include <stdlib.h> 
#include <string.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand_uniform.h>
#include "isingV0.h"

#define RUNS_PER_SIZE 5

int main(int argc, char** argv){
  if(argc!=2){
    printf("Usage: ./v0time [resultFolder]\n");
    exit(1);
  }
  // Get file 
  FILE* result_file;
  char *file_name=(char*)malloc(100*sizeof(char));
  snprintf(file_name,99,"%s/v0time.txt",argv[1]);
  result_file=fopen(file_name,"w");

  // Init variables.
  char **G;
  char **G0;
  int n_min=200;
  int n_step=200;
  int n_max=1000;
  int k_min=20;
  int k_step=20;
  int k_max=100;
  float times[RUNS_PER_SIZE];
  float run_time;
  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // For all sizes until n_max 
  for(int n=n_min;n<=n_max;n+=n_step){
    gridAllocateV0(&G0,n);
    gridAllocateV0(&G,n);
    // For all k until k_max
    for(int k=k_min;k<=k_max;k+=k_step){
      // Test this many times 
      for(int run_count=0;run_count<RUNS_PER_SIZE;run_count++){
        // Run the program:
        hipEventRecord(start,0);
        initRandomV0(&G0,n);
        isingV0(G, G0, n, k);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&run_time,start,stop);
        // Reset events.
        hipEventRecord(start,0);
        hipEventRecord(stop,0);
        // Store time in buffer.
        times[run_count]=run_time;
      }
      // Experiment for (n,k) pair is done: Get median 
      // Ineffiecient code but small size so I don't care.
      float median=0;
      int count;
      for(int i=0;i<RUNS_PER_SIZE;i++){
        count=0;
        for(int j=0;j<RUNS_PER_SIZE;j++){
          if(times[i]>=times[j]){
            count++;
          }
        }
        // Check if median:
        if(count==(RUNS_PER_SIZE/2)+1){
          median=times[i];
          break;
        }
      }
      // Median retrieved write to file:
      // [n] [k] [median]
      fprintf(result_file,"%d %d %f\n",n,k,median);
    }
    // Free grids for next size try.
    freeGridV0(G);
    freeGridV0(G0);
  }
  printf("V0 Timing done.\n");
  // Cleanup.
  hipEventDestroy(start);
  hipEventDestroy(stop);
  free(file_name);
  fclose(result_file);
  return 0; 
}
