#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "isingV3.h"
#include "isingV2.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

int main(){
  // Grids for V3.
  char *G0=NULL;
  char *G=NULL;
  char *HostG=NULL;
  // Grids for V2.
  char **G02=NULL;
  char **G2=NULL;
  // Iteration parameters.
  int n_min=20;
  int n_max=2000;
  int n_step=20;
  int k_min=1;
  int k_max=40;
  int k_step=20;
  // CUDA variables:
  dim3 blockSizeV3,gridSizeV3;
  int blockLength;
  dim3 blockSizeV2,gridSizeV2;
  hipError_t hipError_t;
  // initBlockLength is for the initial state only.
  int initBlockLength=16; 
  int threadBlockSize=2;
  // For each size.. 
  for(int n=n_min;n<=n_max;n+=n_step){
    // Allocate grids for algorithms.
    // V2. 
    gridAllocateV2(&G2,n);
    gridAllocateV2(&G02,n);
    getDimensionsV2(n,blockSizeV2,gridSizeV2,threadBlockSize);
    // V3.
    gridAllocateV3(&G,n);
    gridAllocateV3(&G0,n);
    HostG=(char*)malloc(n*n*sizeof(char));
    // For each iteration count..
    for(int k=k_min;k<=k_max;k+=k_step){
      // Create random state:
      getInitDimensionsV3(n,blockSizeV3,gridSizeV3,initBlockLength);
      initRandomV3<<<gridSizeV3,blockSizeV3>>>(G0,n,initBlockLength);
      hipDeviceSynchronize();
      // Error check for initRandom..
      hipError_t=hipGetLastError();
      if(hipError_t!=hipSuccess){
        printf("Kernel failed at initRandomV3: %s\n",hipGetErrorString(hipError_t));
        exit(1);
      }
      // Hard copy to V2 initial state:
      hipError_t=hipMemcpy(G02[0], G0, n*n*sizeof(char), hipMemcpyDeviceToHost);
      if(hipError_t!=hipSuccess){
        printf("Error at hipMemcpy: %s\n",hipGetErrorString(hipError_t));
        exit(1);
      }
      int ones=0;
      int zeros=0;
      for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
          if(G02[i][j]==1){
            ones++;
          }
          if(G02[i][j]==0){
            zeros++;
          }
        }
      }
      printf("Ones: %d Zeros: %d\n",ones,zeros);
      // Run both algorithms.
      // Grid dimensions update for the second half of the algorithm.
      getIterDimensionsV3(n,blockSizeV3,gridSizeV3,blockLength);
      isingV3(HostG,G,G0,n,k,blockSizeV3,gridSizeV3,blockLength);
      hipDeviceSynchronize();
      isingV2(G2,G02,n,k,blockSizeV2,gridSizeV2,threadBlockSize);
      hipDeviceSynchronize();
      // Compare results and exit if there is an error..
      int errCount=0;
      int bound=0;
      int in=0;
      int *errInd=(int*)malloc(2*(n*n)*sizeof(int));
      for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
          if(HostG[i*n+j]!=G2[i][j]){
            errInd[2*errCount]=i;
            errInd[2*errCount+1]=j;
            errCount++;
          }
        }
      }
      if(errCount>0){
        printf("Results don't match for n:%d and k:%d, error count:%d\n",n,k,errCount);
        for(int i=0;i<errCount;i++){
          printf("(%d,%d)\n",errInd[2*i],errInd[2*i+1]);
        }
        exit(1);
      }
    }
    printf("Size %d done\n",n);
    freeGridV2(G2);
    freeGridV2(G02);
    freeGridV3(G);
    freeGridV3(G0);
    free(HostG);
  }
  printf("Testing successful.\n");
  return 0;

}
